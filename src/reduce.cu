
#include <hip/hip_runtime.h>
template <typename T, unsigned int blockSize, unsigned int dataLength>
__device__ void conditionalWarpReduce(volatile T *sharedData)
{
  if(blockSize >= dataLength)
  {
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

template <typename T, unsigned int blockSize>
__device__ void warpReduce(T* __restrict__ outDataPtr,
    volatile T* __restrict__ sharedData)
{
  conditionalWarpReduce<T, blockSize, 64>(sharedData);
  conditionalWarpReduce<T, blockSize, 32>(sharedData);
  conditionalWarpReduce<T, blockSize, 16>(sharedData);
  conditionalWarpReduce<T, blockSize, 8>(sharedData);
  conditionalWarpReduce<T, blockSize, 4>(sharedData);
  if(threadIdx.x == 0)
    {*outDataPtr= sharedData[0] + sharedData[1];}
  __syncthreads();
}

template <typename T, unsigned int blockSize, unsigned int dataLength>
__device__ void conditionalReduce(volatile T* __restrict__ sharedData)
{
  if(blockSize >= dataLength)
  {
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }

  if((blockSize < dataLength) && (blockSize > (dataLength/2)))
  {
    if(threadIdx.x+(dataLength/2) < blockSize)
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

template <typename T, unsigned int blockSize>
__device__ void blockReduce(T* __restrict__ outGlobalDataPtr,
    volatile T* __restrict__ sharedData)
{
  __syncthreads();
  conditionalReduce<T, blockSize, 1024>(sharedData);
  conditionalReduce<T, blockSize, 512>(sharedData);
  conditionalReduce<T, blockSize, 256>(sharedData);
  conditionalReduce<T, blockSize, 128>(sharedData);

  warpReduce<T, blockSize>(outGlobalDataPtr, sharedData);
  __syncthreads();
}
